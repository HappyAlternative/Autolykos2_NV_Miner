#include "hip/hip_runtime.h"
// prehash.cu

/*******************************************************************************

    PREHASH -- precalculation of hashes

*******************************************************************************/

#include "../include/prehash.h"
#include "../include/compaction.h"
#include "../include/definitions.h"
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
//  Precalculate hashes
////////////////////////////////////////////////////////////////////////////////
int Prehash(
    const int keep,
    // data: pk || mes || w || padding || x || sk
    const uint32_t * data,
    // unfinalized hashes contexts
    uctx_t * uctxs,
    // hashes
    uint32_t * hashes,
    uint32_t  height , 
    uint8_t  AlgVer 
)
{

    if(AlgVer == 1)
    {
        return EXIT_FAILURE ;
    }
    else
    {
        InitPrehash<<<1 + (N_LEN - 1) / BLOCK_DIM, BLOCK_DIM>>>(
            height, hashes
        );
        CUDA_CALL(hipPeekAtLastError());

    }
    return EXIT_SUCCESS;
    
}
__global__ void InitPrehash(
    // height
    const uint32_t  height,
    // hashes
    uint32_t * hashes
)
{
    uint32_t tid = threadIdx.x;

    // shared memory
    __shared__ uint32_t sdata[ROUND_PNP_SIZE_32];

    tid += blockDim.x * blockIdx.x;

    if (tid < N_LEN)
    {
        uint32_t j;

        // local memory
        // 472 bytes
        uint32_t ldata[118];

        // 32 * 64 bits = 256 bytes 
        uint64_t * aux = (uint64_t *)ldata;
        // (212 + 4) bytes 
        ctx_t * ctx = (ctx_t *)(ldata + 64);

        //====================================================================//
        //  Initialize context
        //====================================================================//
        memset(ctx->b, 0, BUF_SIZE_8);
        B2B_IV(ctx->h);
        ctx->h[0] ^= 0x01010000 ^ NUM_SIZE_8;
        memset(ctx->t, 0, 16);
        ctx->c = 0;

        //====================================================================//
        //  Hash tid
        //====================================================================//
#pragma unroll
        for (j = 0; ctx->c < BUF_SIZE_8 && j < INDEX_SIZE_8; ++j)
        {
            ctx->b[ctx->c++] = ((const uint8_t *)&tid)[INDEX_SIZE_8 - j - 1];
        }

        //====================================================================//
        //  Hash height
        //====================================================================//
        #pragma unroll
        for (j = 0; ctx->c < BUF_SIZE_8 && j < HEIGHT_SIZE ; ++j)
        {
            ctx->b[ctx->c++] = ((const uint8_t *)&height)[j];
        }

        //====================================================================//
        //  Hash constant message
        //====================================================================//
#pragma unroll
        for (j = 0; ctx->c < BUF_SIZE_8 && j < CONST_MES_SIZE_8; ++j)
        {
            ctx->b[ctx->c++]
                = (
                    !((7 - (j & 7)) >> 1)
                    * ((j >> 3) >> (((~(j & 7)) & 1) << 3))
                ) & 0xFF;
        }

        while (j < CONST_MES_SIZE_8)
        {
            DEVICE_B2B_H(ctx, aux);

            for ( ; ctx->c < BUF_SIZE_8 && j < CONST_MES_SIZE_8; ++j)
            {
                ctx->b[ctx->c++]
                    = (
                        !((7 - (j & 7)) >> 1)
                        * ((j >> 3) >> (((~(j & 7)) & 1) << 3))
                    ) & 0xFF;
            }
        }

         //====================================================================//
        //  Finalize hash
        //====================================================================//
        DEVICE_B2B_H_LAST(ctx, aux);

#pragma unroll
        for (j = 0; j < NUM_SIZE_8; ++j)
        {
            ((uint8_t *)ldata)[NUM_SIZE_8 - j - 1]
                = (ctx->h[j >> 3] >> ((j & 7) << 3)) & 0xFF;
        }

        //====================================================================//
        //  Dump result to global memory -- BIG ENDIAN
        //====================================================================//
#pragma unroll
        for (int i = 0; i < NUM_SIZE_8-1; ++i) 
        {
            ((uint8_t *)hashes)[tid * NUM_SIZE_8 +i ]
                = ((uint8_t *)ldata)[i];
        }
        ((uint8_t *)hashes)[tid * NUM_SIZE_8 +NUM_SIZE_8-1 ] = 0; 
 
    }

    return;
}
// prehash.cu




